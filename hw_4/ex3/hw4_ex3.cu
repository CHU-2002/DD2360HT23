#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <thrust/device_ptr.h>
#include <thrust/sequence.h>

#define gpuCheck(stmt)                                               \
  do {                                                               \
      hipError_t err = stmt;                                        \
      if (err != hipSuccess) {                                      \
          printf("ERROR. Failed to run stmt %s\n", #stmt);           \
          break;                                                     \
      }                                                              \
  } while (0)

// Macro to check the cuBLAS status
#define cublasCheck(stmt)                                            \
  do {                                                               \
      hipblasStatus_t err = stmt;                                     \
      if (err != HIPBLAS_STATUS_SUCCESS) {                            \
          printf("ERROR. Failed to run cuBLAS stmt %s\n", #stmt);    \
          break;                                                     \
      }                                                              \
  } while (0)

// Macro to check the cuSPARSE status
#define cusparseCheck(stmt)                                          \
  do {                                                               \
      hipsparseStatus_t err = stmt;                                   \
      if (err != HIPSPARSE_STATUS_SUCCESS) {                          \
          printf("ERROR. Failed to run cuSPARSE stmt %s\n", #stmt);  \
          break;                                                     \
      }                                                              \
  } while (0)

 
struct timeval t_start, t_end;
void cputimer_start(){
  gettimeofday(&t_start, 0);
}
void cputimer_stop(const char* info){
  gettimeofday(&t_end, 0);
  double time = (1000000.0*(t_end.tv_sec-t_start.tv_sec) + t_end.tv_usec-t_start.tv_usec);
  printf("Timing - %s. \t\tElasped %.0f microseconds \n", info, time);
}





// Initialize the sparse matrix needed for the heat time step
void matrixInit(double* A, int* ArowPtr, int* AcolIndx, int dimX,
    double alpha) {
  // Stencil from the finete difference discretization of the equation
  double stencil[] = { 1, -2, 1 };
  // Variable holding the position to insert a new element
  size_t ptr = 0;
  // Insert a row of zeros at the beginning of the matrix
  ArowPtr[1] = ptr;
  // Fill the non zero entries of the matrix
  for (int i = 1; i < (dimX - 1); ++i) {
    // Insert the elements: A[i][i-1], A[i][i], A[i][i+1]
    for (int k = 0; k < 3; ++k) {
      // Set the value for A[i][i+k-1]
      A[ptr] = stencil[k];
      // Set the column index for A[i][i+k-1]
      AcolIndx[ptr++] = i + k - 1;
    }
    // Set the number of newly added elements
    ArowPtr[i + 1] = ptr;
  }
  // Insert a row of zeros at the end of the matrix
  ArowPtr[dimX] = ptr;
}

int main(int argc, char **argv) {
  
  cputimer_start();
  int device = 0;            // Device to be used
  int dimX;                  // Dimension of the metal rod
  int nsteps;                // Number of time steps to perform
  double alpha = 0.4;        // Diffusion coefficient
  double beta = 0.0;
  double* temp;              // Array to store the final time step
  double* A;                 // Sparse matrix A values in the CSR format
  int* ARowPtr;              // Sparse matrix A row pointers in the CSR format
  int* AColIndx;             // Sparse matrix A col values in the CSR format
  int nzv;                   // Number of non zero values in the sparse matrix
  double* tmp;               // Temporal array of dimX for computations
  size_t bufferSize = 0;     // Buffer size needed by some routines
  void* buffer = nullptr;    // Buffer used by some routines in the libraries
  int concurrentAccessQ;     // Check if concurrent access flag is set
  double zero = 0;           // Zero constant
  double one = 1;            // One constant
  double norm;               // Variable for norm values
  double error;              // Variable for storing the relative error
  double tempLeft = 200.;    // Left heat source applied to the rod
  double tempRight = 300.;   // Right heat source applied to the rod
  hipblasHandle_t cublasHandle;      // cuBLAS handle
  hipsparseHandle_t cusparseHandle;  // cuSPARSE handle
  hipsparseMatDescr_t Adescriptor;   // Mat descriptor needed by cuSPARSE

  // Read the arguments from the command line
  dimX = atoi(argv[1]);
  nsteps = atoi(argv[2]);
  usePrefetch  = atoi(argv[3]);

  // Print input arguments
  printf("The X dimension of the grid is %d \n", dimX);
  printf("The number of time steps to perform is %d \n", nsteps);

  // Get if the hipDeviceAttributeConcurrentManagedAccess flag is set
  gpuCheck(hipDeviceGetAttribute(&concurrentAccessQ, hipDeviceAttributeConcurrentManagedAccess, device));

  // Calculate the number of non zero values in the sparse matrix. This number
  // is known from the structure of the sparse matrix
  nzv = 3 * dimX - 6;

  //@@ Insert the code to allocate the temp, tmp and the sparse matrix
  //@@ arrays using Unified Memory
  hipMallocManaged(&temp, dimX * sizeof(double));
  hipMallocManaged(&tmp, dimX * sizeof(double));
  hipMallocManaged(&A, nzv * sizeof(double));
  hipMallocManaged(&ARowPtr, (dimX + 1) * sizeof(int));
  hipMallocManaged(&AColIndx, nzv * sizeof(int));



  // Check if concurrentAccessQ is non zero in order to prefetch memory
  if (concurrentAccessQ) {
    //@@ Insert code to prefetch in Unified Memory asynchronously to CPU
    if(usePrefetch != 0){
      
      hipMemPrefetchAsync(temp, dimX * sizeof(double), hipCpuDeviceId, 0);
      hipMemPrefetchAsync(tmp, dimX * sizeof(double), hipCpuDeviceId, 0);
      hipMemPrefetchAsync(A, nzv * sizeof(double), hipCpuDeviceId, 0);
      hipMemPrefetchAsync(ARowPtr, (dimX + 1) * sizeof(int), hipCpuDeviceId, 0);
      hipMemPrefetchAsync(AColIndx, nzv * sizeof(int), hipCpuDeviceId, 0);

    }

  }

  // Initialize the sparse matrix
  matrixInit(A, ARowPtr, AColIndx, dimX, alpha);

  //Initiliaze the boundary conditions for the heat equation
  memset(temp, 0, sizeof(double) * dimX);
  temp[0] = tempLeft;
  temp[dimX - 1] = tempRight;

  if (concurrentAccessQ) {
    //@@ Insert code to prefetch in Unified Memory asynchronously to the GPU
    
    if(usePrefetch != 0){
    hipMemPrefetchAsync(temp, dimX * sizeof(double), device, 0);
    hipMemPrefetchAsync(tmp, dimX * sizeof(double), device, 0);
    hipMemPrefetchAsync(A, nzv * sizeof(double), device, 0);
    hipMemPrefetchAsync(ARowPtr, (dimX + 1) * sizeof(int), device, 0);
    hipMemPrefetchAsync(AColIndx, nzv * sizeof(int), device, 0);
    }
  }

  //@@ Insert code to create the cuBLAS handle
  hipblasCreate(&cublasHandle);    

  //@@ Insert code to create the cuSPARSE handle
  hipsparseCreate(&cusparseHandle);


  //@@ Insert code to set the cuBLAS pointer mode to HIPSPARSE_POINTER_MODE_HOST
  hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST);

  //@@ Insert code to call cusparse api to create the mat descriptor used by cuSPARSE
  hipsparseCreateMatDescr(&Adescriptor);

  //@@ Insert code to call cusparse api to get the buffer size needed by the sparse matrix per
  //@@ vector (SMPV) CSR routine of cuSPARSE
  hipsparseSpMatDescr_t matA;
  hipsparseDnVecDescr_t vecX, vecY;
  void* dBuffer = nullptr;

  // 创建稀疏矩阵描述符
  hipsparseCreateCsr(&matA, dimX, dimX, ARowPtr[dimX], ARowPtr, AColIndx, A, 
                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, 
                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

  // 创建输入和输出向量的描述符
  hipsparseCreateDnVec(&vecX, dimX, temp, HIP_R_64F);
  hipsparseCreateDnVec(&vecY, dimX, tmp, HIP_R_64F);

  // 调用 hipsparseSpMV_bufferSize 获取所需的缓冲区大小
  hipsparseSpMV_bufferSize(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                          &alpha, matA, vecX, &beta, vecY, 
                          HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);


  //@@ Insert code to allocate the buffer needed by cuSPARSE
  hipMalloc(&dBuffer, bufferSize);


  // Perform the time step iterations
  for (int it = 0; it < nsteps; ++it) {
    //@@ Insert code to call cusparse api to compute the SMPV (sparse matrix multiplication) for
    //@@ the CSR matrix using cuSPARSE. This calculation corresponds to:
    //@@ tmp = 1 * A * temp + 0 * tmp
  
    hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);
    //@@ Insert code to call cublas api to compute the axpy routine using cuBLAS.
    //@@ This calculation corresponds to: temp = alpha * tmp + temp
    hipblasDaxpy(cublasHandle, dimX, &alpha, tmp, 1, temp, 1);

    //@@ Insert code to call cublas api to compute the norm of the vector using cuBLAS
    //@@ This calculation corresponds to: ||temp||
    hipblasDnrm2(cublasHandle, dimX, temp, 1, &norm);

    // If the norm of A*temp is smaller than 10^-4 exit the loop
    if (norm < 1e-4)
      break;
  }

  // Calculate the exact solution using thrust
  thrust::device_ptr<double> thrustPtr(tmp);
  thrust::sequence(thrustPtr, thrustPtr + dimX, tempLeft,
      (tempRight - tempLeft) / (dimX - 1));

  // Calculate the relative approximation error:
  one = -1;
  //@@ Insert the code to call cublas api to compute the difference between the exact solution
  //@@ and the approximation
  //@@ This calculation corresponds to: tmp = -temp + tmp
  hipblasDaxpy(cublasHandle, dimX, &one, temp, 1, tmp, 1);

  //@@ Insert the code to call cublas api to compute the norm of the absolute error
  //@@ This calculation corresponds to: || tmp ||
  hipblasDnrm2(cublasHandle, dimX, tmp, 1, &norm);

  error = norm;
  //@@ Insert the code to call cublas api to compute the norm of temp
  //@@ This calculation corresponds to: || temp ||
  hipblasDnrm2(cublasHandle, dimX, temp, 1, &norm);

  // Calculate the relative error
  error = error / norm;
  printf("The relative error of the approximation is %f\n", error);

  //@@ Insert the code to destroy the mat descriptor
  hipsparseDestroyMatDescr(Adescriptor);

  //@@ Insert the code to destroy the cuSPARSE handle
  hipsparseDestroy(cusparseHandle);

  //@@ Insert the code to destroy the cuBLAS handle
  hipblasDestroy(cublasHandle);


  //@@ Insert the code for deallocating memory
  hipFree(temp);
  hipFree(tmp);
  hipFree(A);
  hipFree(ARowPtr);
  hipFree(AColIndx);

  
    if(usePrefetch == 0){
    cputimer_stop("Total Execuation TIme without Prefetch");

    }else{
      
    cputimer_stop("Total Execuation TIme with Prefetch");
    }
  return 0;
}
